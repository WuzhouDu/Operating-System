﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */

  /* Searching the file name in FCB */
  int found = 0;
  u32 fp;
  for (int i = 0; i < fs->FCB_ENTRIES; i++){
    if (!found){
      for (int j = 0; j < fs->FCB_SIZE; j++){
        if (s[j] != fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j]) break;
        else if ((s[j] == fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j]) && (s[j] == '\0')) {
          // if found the file, return
          found = 1;
          fp = fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(((fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 20]) << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 21]);
          // printf("[fs_open] : Open existing file: %s, fp is %x, FCB = %d, start block index = %x, length = %d, time = %d\n", 
            // s, fp, i,
            // (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 21],
            // (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 23],
            // (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 25]);
          return fp;
        }
      }
    }
  }

  if ((!found) && (op == G_READ)) {
    printf("[fs_open ERROR] : G_READ nonexisting file: %s. RETURN.\n", s);
    return 0XFFFFFFFF;
  }
  if ((!found) && (op == G_WRITE)){
    // printf("[fs_open] : G_WRITE new file: %s.\n", s);
  
  /* Handle the G_WRITE new file. */

    /* traverse the VCB to record the first available block */
    int first_avai_block = 0xffff;
    for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++){
      if (first_avai_block != 0xffff) break;
      if (fs->volume[i] != 0xff){
        for (int j = 7; j >= 0; j--){
          if (((fs->volume[i] & (1 << j)) >> j) == 0) {
            // record this block
            first_avai_block = i*8 + (7-j);

            // write to VCB to allocate memory.
            fs->volume[i] += (1 << j);
            // printf("[fs_open] : find available block %u\n", first_avai_block);
            // printf("[fs_open] : update VCB %d to %x\n", i, fs->volume[i]);
            break;
          }
        }
      }
    }
    if (first_avai_block == 0xffff) {
      printf("[ERROR fs_open] : no available block.\n");
      return 0XFFFFFFFF;
    }

    /* traverse the FCB to find first available FCB*/
    int first_avai_FCB = 0xfff;
    for (int i = 0; i < fs->FCB_ENTRIES; i++){
      if (first_avai_FCB == 0xfff){
        for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){ // browse the file name of this FCB to judge whether available.
          if (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j] != 0) break;
          if (j == fs->MAX_FILENAME_SIZE - 1) {
            first_avai_FCB = i;
            // printf("[fs_open] : find available FCB %x\n", first_avai_FCB);
            break;
          }
        }
      }
      else break;
    }

    if (first_avai_FCB == 0xfff) {
      printf("[ERROR fs_open] : no available FCB.\n");
      return 0XFFFFFFFF;
    }
    
    /* write the meta data to this available FCB */
    // write the file name.
    for (int i = 0; i < fs->MAX_FILENAME_SIZE; i++){
      fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + i] = s[i];
      if (s[i] == '\0') break;
    }
    
    // write the start point, length = 0, time and create time.
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 20] = (first_avai_block >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 21] = (first_avai_block & 0xff);
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 24] = (gtime >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 25] = (gtime & 0xff);
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 26] = (gtime >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 27] = (gtime & 0xff);    
    
    fp = fs->FILE_BASE_ADDRESS + ((fs->STORAGE_BLOCK_SIZE)*(((fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 20]) << 8) + fs->volume[fs->SUPERBLOCK_SIZE + first_avai_FCB*(fs->FCB_SIZE) + 21]));
    // printf("[fs_open] : write meta data to FCB %x, name = %s, time = %x, length = 0, start address = %x\n", first_avai_FCB, s, gtime, fp);
    gtime++;
    return fp;
  }
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	for (int i = 0; i < size; i++){
    output[i] = fs->volume[fp + i];
    // printf("[fs_read] : read %x\n", output[i]);
   
  }
  // printf("[fs_read] : fs_read\n");
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	int block_index = (fp - fs->FILE_BASE_ADDRESS)/fs->STORAGE_BLOCK_SIZE;
  int write_block_size = (size == 0) ? 1 : 1 + (size-1) / fs->STORAGE_BLOCK_SIZE;
  // int write_block_size = size / fs->STORAGE_BLOCK_SIZE;
  int older_block_size = 0;
  int older_length = 0;
  int older_create_time;
  int FCB_index = 0xfff;
  uchar name[20];

  /* traverse the FCB to get the metadata of this file */
  for (int i = 0; i < fs->FCB_ENTRIES; i++){
    if (((fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 21]) == block_index){
      FCB_index = i;
      older_length = (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 23];
      older_block_size = (older_length == 0) ? 1 : 1 + (older_length-1) / fs->STORAGE_BLOCK_SIZE;
      older_create_time = (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 26] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 27];
      // older_block_size = older_length / fs->STORAGE_BLOCK_SIZE;
      
      
      for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){
        name[j] = fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j];
        if (name[j] == '\0') break;
      }
      // printf("[fs_write] : find older file. FCB = %d, name = %s, start block index %x, length = %d, time = %d\n", 
        // FCB_index, name, block_index, 
        // (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 23],
        // (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 25]);
      break;
    }
  }
  if (FCB_index == 0xfff) {
    printf("[ERROR fs_write] : nonexist file in FCB\n");
    return 0xffffffff;
  }

  // if the original block size is the same as the new write block size, the block index will not change
  if (older_block_size == write_block_size){

    // rewrite the file content
    for (int i = 0; i < write_block_size*(fs->STORAGE_BLOCK_SIZE); i++){
      fs->volume[fs->FILE_BASE_ADDRESS + block_index*(fs->STORAGE_BLOCK_SIZE) + i] = (i < size) ? input[i] : 0;
    }

    // rewrite the FCB, only length and time need to rewrite
    fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 22] = (size >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 23] = (size & 0xff);
    fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 24] = (gtime >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 25] = (gtime & 0xff);
    // printf("[fs_write] : lucky dog! write file is the same block size as old file. write length = %d\n", size);
    // printf("暂时看VCB[0] VCB[1]: %x %x\n", fs->volume[0], fs->volume[1]);
    gtime++;
    return 0;
  }

  /* else if original block size is different from the new write block size */
  else{
    // search next FCB's name to see whether this is the last file
    int last_file = 1;
    for (int i = 0; i < fs->MAX_FILENAME_SIZE; i++){
      if (last_file) {
        if (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + 1)*(fs->FCB_SIZE) + i] != 0){
          last_file = 0;
          break;
        }
      }
    }

    // if this file is the last file. no need to compact
    if (last_file || (FCB_index == fs->FCB_ENTRIES - 1)) {
      
      if (older_length <= size) {
        // rewrite the file content
        for (int i = 0; i < size; i++){
          fs->volume[fs->FILE_BASE_ADDRESS + block_index*(fs->STORAGE_BLOCK_SIZE) + i] = input[i];
        }
      }
      else {
        // rewrite the file content
        for (int i = 0; i < older_length; i++){
          fs->volume[fs->FILE_BASE_ADDRESS + block_index*(fs->STORAGE_BLOCK_SIZE) + i] = (i < size) ? input[i] : 0;
        }
      }
      
      // rewrite the FCB, only length and time need to rewrite
      fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 22] = (size >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 23] = (size & 0xff);
      fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 24] = (gtime >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + FCB_index*(fs->FCB_SIZE) + 25] = (gtime & 0xff);

      // update VCB
      int VCB_index;
      int VCB_offset;
      for (int i = 0; i < write_block_size; i++){
        VCB_index = (i + block_index) / 8;
        VCB_offset = (i + block_index) % 8;
        fs->volume[VCB_index] |= (1 << (7 - VCB_offset));
        if (i == write_block_size - 1){
          if (write_block_size < older_block_size) {
            for (int j = 0; j < older_block_size - write_block_size; j++){
              VCB_index = (i + block_index+1 + j) / 8;
              VCB_offset = (i + block_index+1 + j) % 8;
              fs->volume[VCB_index] &= (0xff - (1 << (7 - VCB_offset))); 
            }
          }
        }
      }

      // printf("[fs_write] : write file is the last file as before. write from block %d, length = %d\n", block_index, size);
      // printf("暂时看VCB[0] VCB[1]: %x %x\n", fs->volume[0], fs->volume[1]);
      gtime++;
      return 0;
    }

    // block size different, and is not the last file. need to compact all the files after this file, then write to an available FCB
    
    /* do the compaction */
    else {
      int new_avai_FCB = fs->MAX_FILE_NUM - 1;
      int new_avai_block = (1 << 15) - 1;

      // compact the contents of file
      for (int i = 0; i < (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE); i++){
        int VCB_index = (block_index + i) / 8;
        int VCB_offset = (block_index + i) % 8;
        if ((fs->volume[VCB_index] & (1 << (7 - VCB_offset))) != 0){
          for (int j = 0; j < fs->STORAGE_BLOCK_SIZE; j++){
            if (block_index + i +older_block_size >= (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE)) {
              fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i) + j] = 0;
            }
            else fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i) + j] = fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i +older_block_size) + j];
          }
        }
        else {
          new_avai_block = block_index + i - older_block_size;
          break;
        }
      }
      


      // compact the VCB (from the backward direction)
      // find the last allocated block
      int last_block = (1 << 15);
      for (int i = 0; i < (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE); i++){
        int VCB_index = i / 8;
        int VCB_offset = i % 8;
        if ((fs->volume[VCB_index] & (1 << (7-VCB_offset))) == 0) {
          last_block = i - 1;
          // printf("write compact: 最后一个block index为 %d\n", last_block);
          break;
        }
      }
      
      for (int i = 0; i < older_block_size; i++){
        int VCB_index = (last_block - i) / 8;
        int VCB_offset = (last_block - i) % 8;
        fs->volume[VCB_index] &= (0xff - (1 << (7 - VCB_offset)));
        // printf("压VCB: VCB[0], VCB[1]: %x %x\n", fs->volume[0], fs->volume[1]);
      }
      

      // compact the FCB
      for (int i = 0; i < fs->FCB_ENTRIES; i++){
        if (FCB_index + i + 1 >= fs->FCB_ENTRIES) {
          for (int j = 0; j < fs->FCB_SIZE; j++){
            fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = 0;
          }
          break;
        }
        int vacant_next_FCB = 1;
        for (int j = 0; j < 20; j++){
          if (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + 1+i)*(fs->FCB_SIZE) + j] != 0) {
            vacant_next_FCB = 0;
            break;
          }
        }
        if (vacant_next_FCB){
          for (int j = 0; j < fs->FCB_SIZE; j++){
            fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = 0;
          }
          new_avai_FCB = i;
          break;
        }
        int prev_start_block = (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + 21];
        int new_start_block = prev_start_block - older_block_size;
        
        // update name
        for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){
          fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i + 1)*(fs->FCB_SIZE) + j];
        }

        // update start block index
        fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 20] = (new_start_block >> 8);
        fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 21] = (new_start_block & 0xff);
        
        // update length, time and create time
        for (int j = 22; j < 28; j++){
          fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + j];
        }
      
      }

      /* compaction finished. */
      // start write new file.

      // write to VCB
      // printf("7\n");
      int new_VCB_index;
      int new_VCB_offset;
      for (int i = 0; i < write_block_size; i++){
        new_VCB_index = (i + new_avai_block) / 8;
        new_VCB_offset = (i + new_avai_block) % 8;
        fs->volume[new_VCB_index] |= (1 << (7 - new_VCB_offset));
        // printf("写VCB: VCB[0], VCB[1]: %x %x\n", fs->volume[0], fs->volume[1]);
      }
      
      
      // write to FCB
      for (int i = 0; i < fs->MAX_FILENAME_SIZE; i++) {
        // printf("name[%d] = %c\n", i, name[i]);
        // printf("8\n");
        // printf("new_avai_FCB = %x\n", new_avai_FCB);
        fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + i] = name[i];
        if (name[i] == '\0') {
          break;
        }
      }
      
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 20] = (new_avai_block >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 21] = (new_avai_block & 0xff);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 22] = (size >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 23] = (size & 0xff);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 24] = (gtime >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 25] = (gtime & 0xff);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 26] = (older_create_time >> 8);
      fs->volume[fs->SUPERBLOCK_SIZE + new_avai_FCB*(fs->FCB_SIZE) + 27] = (older_create_time & 0xff);
      

      // write to file content
      for (int i = 0; i < size; i++){
        fs->volume[fs->FILE_BASE_ADDRESS + new_avai_block*(fs->STORAGE_BLOCK_SIZE) + i] = input[i];
      }

      // printf("[fs_write] : write between files. write from block %d, length = %d\n", new_avai_block, size);
      // printf("暂时看VCB[0] VCB[1]: %x %x\n", fs->volume[0], fs->volume[1]);
      gtime++;
      return 0;
    }
  }
}

__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
  if (op == LS_D){
    // sort by modified time
    int prev_max;
    int now_max;
    int now_max_index;
    
    printf("[fs_gsys] : ===sort by modified time===\n");
    for (int i = 0; i < fs->FCB_ENTRIES; i++){
      int vacant = 1;
      for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
        if (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + n] != 0){
          vacant = 0;
          // printf("没有空啊\n");
          break;
        }
      }
      if (vacant) return;
      int complete = 1;
      // find minimum time.
      if (i == 0) prev_max = 2147483647;
      for (int j = 0; j < fs->FCB_ENTRIES; j++){
        int vacant = 1;
        for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
          if (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + n] != 0){
            vacant = 0;
            // printf("没有空啊\n");
            break;
          }
        }
        if (vacant) break;

        int time = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 25];
        if (time < prev_max){
          now_max = time;
          now_max_index = j;
          complete = 0;
          // printf("这里time为%d, index为%d\n", now_max, now_max_index);
          break;
        }
      }
      if (complete) return;

      for (int j = 0; j < fs->FCB_ENTRIES; j++){
        // make sure this is not vacant;
        int vacant = 1;
        for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
          if (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + n] != 0){
            vacant = 0;
            // printf("没有空啊\n");
            break;
          }
        }
        if (vacant) break;
        int time = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 25];
        if ((time < prev_max) && (time > now_max)){
          now_max = time;
          now_max_index = j;
        }
      }

      char name[20];
      for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){
        name[j] = fs->volume[fs->SUPERBLOCK_SIZE + now_max_index*(fs->FCB_SIZE) + j];
        if (name[j] == '\0') break;
      }
      printf("[fs_gsys] : %s\n", name);
      prev_max = now_max;
    }
  }
  else if (op == LS_S){
    // list by size, same size -> first create first print
    int prev_max_size;
    int prev_min_time;
    int now_max_size;
    int now_min_time;
    int now_max_index;

    printf("[fs_gsys] : ===sort by file size===\n");
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      int vacant = 1;
      for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
        if (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + n] != 0){
          vacant = 0;
          // printf("没有空啊\n");
          break;
        }
      }
      if (vacant) return;

      int complete = 1;
      if (i == 0) {
        prev_max_size = 0xffff;
        prev_min_time = -1;
      }
      for (int j = 0; j < fs->FCB_ENTRIES; j++) {
        int vacant = 1;
        for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
          if (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + n] != 0){
            vacant = 0;
            // printf("没有空啊\n");
            break;
          }
        }
        if (vacant) break;

        int size = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 23];
        int create_time = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 26] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 27];
        if ((size < prev_max_size) || ((size == prev_max_size) && (create_time > prev_min_time))) {
          now_max_size = size;
          now_max_index = j;
          now_min_time = create_time;
          complete = 0;
          // printf("[fs_gsys LS] : found at least one candidate at %dth FCB. now_min_size = %d, now_min_time = %d\n", j, now_min_size, now_min_time);
          break;
        }
      }

      if (complete) {
        // printf("[fs_gsys LS] : LS_S complete.\n");
        return;
      }

      for (int j = 0; j < fs->FCB_ENTRIES; j++){
        int vacant = 1;
        for (int n = 0; n < fs->MAX_FILENAME_SIZE; n++){
          if (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + n] != 0){
            vacant = 0;
            // printf("没有空啊\n");
            break;
          }
        }
        if (vacant) break;
        int size = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 23];
        int create_time = (fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 26] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + j*(fs->FCB_SIZE) + 27];
        if ((size < prev_max_size) || ((size == prev_max_size) && (create_time > prev_min_time))) {
          if ((size > now_max_size) || ((size == now_max_size) && (create_time < now_min_time))){
            now_max_size = size;
            now_max_index = j;
            now_min_time = create_time;
          }
        }
      }

      prev_max_size = now_max_size;
      prev_min_time = now_min_time;
      char name[20];
      for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){
        name[j] = fs->volume[fs->SUPERBLOCK_SIZE + now_max_index*(fs->FCB_SIZE) + j];
        if (name[j] == '\0') break;
      }
      printf("[fs_gsys] : %s %d\n", name, now_max_size);
    }
  }
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
  /* search file name in FCB */
  int found = 0;
  u32 fp;
  int FCB_index;
  int block_index;
  int length;
  int block_size;

  if (op != RM) {
    printf("[fs_gsys ERROR] : unkonwn operation.\n");
    return;
  }
  
  // printf("REMOVE前, VCB[4092]: %x\n", fs->volume[4092]);

  for (int i = 0; i < fs->FCB_ENTRIES; i++){
    if (!found){
      for (int j = 0; j < fs->FCB_SIZE; j++){
        if (s[j] != fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j]) break;
        else if ((s[j] == fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + j]) && (s[j] == '\0')) {
          // if found the file, return
          found = 1;
          FCB_index = i;
          block_index = (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 21];
          length = (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 23];
          block_size = (length == 0) ? 1 : (1 + (length-1) / fs->STORAGE_BLOCK_SIZE);
          fp = fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(((fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 20]) << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 21]);
          // printf("[fs_gsys REMOVE] : Remove existing file: %s, fp is %x, FCB = %d, start block index = %x, length = %d, time = %d\n", 
          //   s, fp, i, block_index, length,
          //   (fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + i*(fs->FCB_SIZE) + 25]);
          break;
        }
      }
    }
    else break;
  }
  
  /* do the compaction */

  // compact the contents of file
  for (int i = 0; i < fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE; i++){
    if (block_index + i == (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE)) break;
    int VCB_index = (block_index + i) / 8;
    int VCB_offset = (block_index + i) % 8;
    if ((fs->volume[VCB_index] & (1 << (7 - VCB_offset))) != 0){

      for (int j = 0; j < fs->STORAGE_BLOCK_SIZE; j++){
        if (block_index + i +block_size >= (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE)) {
          fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i) + j] = 0;
        }
        else fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i) + j] = fs->volume[fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE*(block_index + i +block_size) + j];
      }
    }
    else break;
  }


  // compact the VCB (from the backward direction)
  int last_block = (1 << 15);
  for (int i = 0; i < (fs->MAX_FILE_SIZE / fs->STORAGE_BLOCK_SIZE); i++){
    int VCB_index = i / 8;
    int VCB_offset = i % 8;
    if ((fs->volume[VCB_index] & (1 << (7-VCB_offset))) == 0) {
      last_block = i - 1;
      // printf("最后一个block index为 %d\n", last_block);
      break;
    }
  }
  for (int i = 0; i < block_size; i++){
    int VCB_index = (last_block - i) / 8;
    int VCB_offset = (last_block - i) % 8;
    fs->volume[VCB_index] &= (0xff - (1 << (7 - VCB_offset)));
    // printf("压VCB: VCB[%d]: %x\n", VCB_index, fs->volume[VCB_index]);
  }

  // compact the FCB
  for (int i = 0; i < fs->FCB_ENTRIES; i++){
    // if (FCB_index + i == fs->FCB_ENTRIES) break;
    
    if (FCB_index + i + 1 == fs->FCB_ENTRIES) {
      for (int j = 0; j < fs->FCB_SIZE; j++){
        fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = 0;
      }
      break;
    }

    int vacant_next_FCB = 1;
    for (int j = 0; j < 20; j++){
      if (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + 1+i)*(fs->FCB_SIZE) + j] != 0) {
        vacant_next_FCB = 0;
        break;
      }
    }
    if (vacant_next_FCB){
      for (int j = 0; j < fs->FCB_SIZE; j++){
        fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = 0;
      }
      break;
    }
    int prev_start_block = (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + 21];
    int new_start_block = prev_start_block - block_size;
    // printf("FCB index = %d, i = %d, prev_start_block = %x, new_start_block = %x \n", FCB_index, i, prev_start_block, new_start_block);
    // update name
    char new_name[20];
    for (int j = 0; j < fs->MAX_FILENAME_SIZE; j++){
      fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i + 1)*(fs->FCB_SIZE) + j];
      new_name[j] = fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j];
    }

    // update start block index
    fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 20] = (new_start_block >> 8);
    fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 21] = (new_start_block & 0xff);

    // update length, time and create time
    for (int j = 22; j < 28; j++){
      fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + j] = fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i+1)*(fs->FCB_SIZE) + j];
    }
    // printf("Update file: name = %s, FCB = %d, start block index = %x, length = %d, time = %d\n", 
    //   new_name, FCB_index + i,
    //   (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 20] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 21],
    //   (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 22] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 23],
    //   (fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 24] << 8) + fs->volume[fs->SUPERBLOCK_SIZE + (FCB_index + i)*(fs->FCB_SIZE) + 25]);
  }
  
  // printf("REMOVE后, VCB[4092]: %x\n", fs->volume[4092]);
}